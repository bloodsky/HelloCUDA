#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define VECT_SIZE 10000000
#define THRESHOLD 1.e-7

__global__ void VectorAdd(float *w, float *u, float *v, int n) {
    for (int i = 0; i < n; i++) {
        w[i] = u[i] + v[i];
    }
}

void fill(float *v) {
    for (int i = 0; i < VECT_SIZE; i++) {
        v[i] = (float)i;
    }
}

int main() {

    // Host pointers
    float *u, *v, *w;
    // Device pointers
    float *u_device, *v_device, *w_device;

    // Alloco mem host
    u = (float *) malloc(sizeof(float)*VECT_SIZE);
    v = (float *) malloc(sizeof(float)*VECT_SIZE);
    w = (float *) malloc(sizeof(float)*VECT_SIZE);

    fill(u);
    fill(v);

    // Alloco mem device 
    hipMalloc((void **) &u_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &v_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &w_device, sizeof(float)*VECT_SIZE);

    // Trasferisco i dati dall'host area alla device area
    hipMemcpy(u_device, u, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    hipMemcpy(v_device, v, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    
    // Kernel call
    VectorAdd<<<1,1>>>(w_device, u_device, v_device, VECT_SIZE);

    // Copio dal risultato ottenuto sulla GPU nell'area dell'host
    hipMemcpy(w, w_device, sizeof(float)*VECT_SIZE, hipMemcpyDeviceToHost);

    // check --> 3 - 1 - 2 < 0.0000001 ?  
    for (int i = 0; i < VECT_SIZE; i++) {
        if (!((w[i]-u[i]-v[i]) < THRESHOLD)) {
            fprintf(stderr,"Got mistake!\n");
        }
    }

    hipFree(u_device);
    hipFree(v_device);
    hipFree(w_device);

    free(u);
    free(v);
    free(w);
}
