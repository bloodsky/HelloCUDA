#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define VECT_SIZE 10000000
#define THRESHOLD 1.e-7

__global__ void VectorAdd(float *w, float *u, float *v, int n) {
    
    int index = threadIdx.x;
    int step = blockDim.x;

    for (int i = index; i < n; i += step) {
        w[i] = u[i] + v[i];
    }
}

void fill(float *v) {
    for (int i = 0; i < VECT_SIZE; i++) {
        v[i] = (float)i;
    }
}

int main() {

    // Host pointers
    float *u, *v, *w;
    // Device pointers
    float *u_device, *v_device, *w_device;

    // Alloco mem host
    u = (float *) malloc(sizeof(float)*VECT_SIZE);
    v = (float *) malloc(sizeof(float)*VECT_SIZE);
    w = (float *) malloc(sizeof(float)*VECT_SIZE);

    fill(u);
    fill(v);

    // Alloco mem device 
    hipMalloc((void **) &u_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &v_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &w_device, sizeof(float)*VECT_SIZE);

    // H --> D
    hipMemcpy(u_device, u, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    hipMemcpy(v_device, v, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    
    // Kernel call
    VectorAdd<<<1,256>>>(w_device, u_device, v_device, VECT_SIZE);

    // D --> H
    hipMemcpy(w, w_device, sizeof(float)*VECT_SIZE, hipMemcpyDeviceToHost);

    // check ?
    for (int i = 0; i < VECT_SIZE; i++) {
        if (!((w[i]-u[i]-v[i]) < THRESHOLD)) {
            fprintf(stderr,"Got mistake!\n");
        }
    }

    hipFree(u_device);
    hipFree(v_device);
    hipFree(w_device);

    free(u);
    free(v);
    free(w);
}
