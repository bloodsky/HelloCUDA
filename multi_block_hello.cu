#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define VECT_SIZE 10000000
#define THRESHOLD 1.e-7
#define BLOCK_SIZE 256

__global__ void VectorAdd(float *w, float *u, float *v, int n) {
    
    // starting point for each blocks
    int SPFB = blockIdx.x * blockDim.x;
    int TID = SPFB + threadIdx.x;  

    // range checking
    if (TID < n) w[TID] = u[TID] + v[TID];
}

void fill(float *v) {
    for (int i = 0; i < VECT_SIZE; i++) {
        v[i] = (float)i;
    }
}

int main() {

    // Host pointers
    float *u, *v, *w;
    // Device pointers
    float *u_device, *v_device, *w_device;

    // Alloco mem host
    u = (float *) malloc(sizeof(float)*VECT_SIZE);
    v = (float *) malloc(sizeof(float)*VECT_SIZE);
    w = (float *) malloc(sizeof(float)*VECT_SIZE);

    fill(u);
    fill(v);

    // Alloco mem device :
    hipMalloc((void **) &u_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &v_device, sizeof(float)*VECT_SIZE);
    hipMalloc((void **) &w_device, sizeof(float)*VECT_SIZE);

    // Trasferisco i dati dall'host area alla device area
    hipMemcpy(u_device, u, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    hipMemcpy(v_device, v, sizeof(float)*VECT_SIZE, hipMemcpyHostToDevice); 
    
    // Kernel call
    int num_blocks = (VECT_SIZE/BLOCK_SIZE) + 1;
    VectorAdd<<<num_blocks,256>>>(w_device, u_device, v_device, VECT_SIZE);

    // Copio dal risultato ottenuto sulla GPU nell'area dell'host
    hipMemcpy(w, w_device, sizeof(float)*VECT_SIZE, hipMemcpyDeviceToHost);

    // correct threshold?  
    for (int i = 0; i < VECT_SIZE; i++) {
        if (!((w[i]-u[i]-v[i]) < THRESHOLD)) {
            fprintf(stderr,"Got mistake!\n");
        }
    }

    hipFree(u_device);
    hipFree(v_device);
    hipFree(w_device);

    free(u);
    free(v);
    free(w);
}
